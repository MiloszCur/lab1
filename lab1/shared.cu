#include "hip/hip_runtime.h"
/*
CUDA - prepare the histogram of N numbers in range of <a;b> where a and b should be integers
*/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__host__ void errorexit(const char *s)
{
    printf("\n%s", s);
    exit(EXIT_FAILURE);
}

__global__ void computeHistogramSharedMemory(int *data, int *globalHistogram, int N, int A, int B)
{
    // Declare shared memory for the histogram
    extern __shared__ int sharedHistogram[];

    // Initialize shared memory histogram to 0
    int threadId = threadIdx.x;
    if (threadId < (B - A))
    {
        sharedHistogram[threadId] = 0;
    }
    __syncthreads();

    // Calculate global thread ID
    int globalId = blockIdx.x * blockDim.x + threadId;

    // Process data points in this block
    if (globalId < N)
    {
        int resultIdx = (data[globalId] - A);
        atomicAdd(&sharedHistogram[resultIdx], 1);
    }
    __syncthreads();

    // Write shared histogram back to global memory
    if (threadId < (B - A))
    {
        atomicAdd(&globalHistogram[threadId], sharedHistogram[threadId]);
    }
}

void generateRandomNumbers(int *arr, int N, int A, int B)
{

    srand(time(NULL));

    for (int i = 0; i < N; i++)
    {
        arr[i] = A + rand() % (B - A + 1);
    }
}

int main(int argc, char **argv)
{

    int threadsinblock = 1024;
    int blocksingrid;

    int N, A, B;
    hipEvent_t start, stop;
    float milliseconds = 0;

    printf("Enter number of elements: \n");
    scanf("%d", &N);

    printf("Enter A value (start range): \n");
    scanf("%d", &A);

    printf("Enter B value (end range): \n");
    scanf("%d", &B);

    int *randomNumbers = (int *)malloc(N * sizeof(int));
    if (randomNumbers == NULL)
    {
        printf("Memory allocation failed.\n");
        return 1;
    }

    generateRandomNumbers(randomNumbers, N, A, B);

    blocksingrid = ceil((double)N / threadsinblock);

    printf("The kernel will run with: %d blocks\n", blocksingrid);

    int *resultArrayHost, *resultArrayDevice, *randomNumbersDevice;

    resultArrayHost = (int *)calloc((B - A), sizeof(int));

    if (resultArrayHost == NULL)
    {
        printf("Memory allocation failed.\n");
        return 1;
    }

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipMalloc((void **)&randomNumbersDevice, N * sizeof(int));
    hipMalloc((void **)&resultArrayDevice, (B - A) * sizeof(int));

    hipMemcpy(randomNumbersDevice, randomNumbers, N * sizeof(int), hipMemcpyHostToDevice);

    // Initialize device histogram to 0
    hipMemset(resultArrayDevice, 0, (B - A) * sizeof(int));

    int sharedMemorySize = (B - A) * sizeof(int);

    computeHistogramSharedMemory<<<blocksingrid, threadsinblock, sharedMemorySize>>>(randomNumbersDevice, resultArrayDevice, N, A, B);

    // Copy the histogram result back to the host
    hipMemcpy(resultArrayHost, resultArrayDevice, (B - A) * sizeof(int), hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);

    // Wait for the stop event to finish
    hipEventSynchronize(stop);

    // Calculate elapsed time
    hipEventElapsedTime(&milliseconds, start, stop);

    // Print the histogram
    printf("Histogram:\n");
    for (int i = 0; i < B - A; i++)
    {
        printf("%d occures %d\n", i, resultArrayHost[i]);
    }

    printf("Kernel execution time: %.3f ms\n", milliseconds);
    // Free allocated memory
    free(randomNumbers);
    free(resultArrayHost);
    hipFree(randomNumbersDevice);
    hipFree(resultArrayDevice);

    return 0;
}

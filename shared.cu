#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__host__ void errorexit(const char *s)
{
    printf("\n%s", s);
    exit(EXIT_FAILURE);
}

__global__ void computeSumAndAverageShared(int *data, int N, int *sum)
{
    // Deklaracja pamięci dzielonej
    extern __shared__ int sharedSum[];

    int threadId = threadIdx.x;
    int globalId = blockIdx.x * blockDim.x + threadId;

    // Inicjalizowanie pamięci dzielonej
    if (threadId == 0)
    {
        sharedSum[0] = 0;
    }
    __syncthreads();

    // Jeśli wątek jest w zakresie danych
    if (globalId < N)
    {
        atomicAdd(&sharedSum[0], data[globalId]); // Dodaj do sumy w pamięci dzielonej
    }
    __syncthreads();

    // Przepisanie sumy do globalnej pamięci
    if (threadId == 0)
    {
        atomicAdd(sum, sharedSum[0]);
    }
}

void generateRandomNumbers(int *arr, int N, int A, int B)
{
    srand(time(NULL));
    for (int i = 0; i < N; i++)
    {
        arr[i] = A + rand() % (B - A + 1);
    }
}

int main(int argc, char **argv)
{
    int threadsinblock = 1024;
    int blocksingrid;

    int N, A, B;
    hipEvent_t start, stop;
    float milliseconds = 0;

    printf("Enter the number of elements: \n");
    scanf("%d", &N);

    printf("Enter A value (start range): \n");
    scanf("%d", &A);

    printf("Enter B value (end range): \n");
    scanf("%d", &B);

    int *randomNumbers = (int *)malloc(N * sizeof(int));
    if (randomNumbers == NULL)
    {
        printf("Memory allocation failed.\n");
        return 1;
    }

    generateRandomNumbers(randomNumbers, N, A, B);

    blocksingrid = (N + threadsinblock - 1) / threadsinblock;

    printf("The kernel will run with: %d blocks\n", blocksingrid);

    int *sumHost, *sumDevice;
    sumHost = (int *)malloc(sizeof(int));
    *sumHost = 0;

    int *randomNumbersDevice;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipMalloc((void **)&randomNumbersDevice, N * sizeof(int));
    hipMalloc((void **)&sumDevice, sizeof(int));

    hipMemcpy(randomNumbersDevice, randomNumbers, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(sumDevice, 0, sizeof(int)); // Inicjalizuj sumę na GPU

    // Rozmiar pamięci dzielonej
    int sharedMemorySize = sizeof(int);

    computeSumAndAverageShared<<<blocksingrid, threadsinblock, sharedMemorySize>>>(randomNumbersDevice, N, sumDevice);

    // Kopiuj wynik sumy do hosta
    hipMemcpy(sumHost, sumDevice, sizeof(int), hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    // Oblicz średnią
    float average = (float)(*sumHost) / N;

    printf("Sum: %d\n", *sumHost);
    printf("Average: %.2f\n", average);
    printf("Kernel execution time: %.3f ms\n", milliseconds);

    // Free allocated memory
    free(randomNumbers);
    free(sumHost);
    hipFree(randomNumbersDevice);
    hipFree(sumDevice);

    return 0;
}
